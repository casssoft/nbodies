#include "hip/hip_runtime.h"
#include <memory>
#include <vector>

#include "Particle.h"


void stepParticles(Particles &particles, double step, double softening) {

  //for (unsigned int i = 0; i < particles.length; ++i) {
  //  // Not actually acceleration
  //  // This is the acceleration accumulator
  //  //particles[i]->setAcceleration(Vector3d::Zero());
  //  particles.acceleration[i*3] = 0;
  //  particles.acceleration[i*3 + 1] = 0;
  //  particles.acceleration[i*3 + 2] = 0;
  //}

  for (unsigned int i = 0; i < particles.length; ++i) {
    for (unsigned int j = i + 1; j < particles.length; ++j) {
      // r_i_j is the distance vector
      //Vector3d r_i_j = particles[j]->getPosition() - particles[i]->getPosition();
      double r_i_j_x = particles.positions.xs[j] - particles.positions.xs[i];
      double r_i_j_y = particles.positions.ys[j] - particles.positions.ys[i];
      double r_i_j_z = particles.positions.zs[j] - particles.positions.zs[i];

      // bottom is scaling factor we divide by, we don't need to separate it out
      //double bottom = r_i_j.squaredNorm() + e2;
      double bottom = r_i_j_x * r_i_j_x + r_i_j_y * r_i_j_y + r_i_j_z * r_i_j_z + softening;

      bottom = sqrt(bottom * bottom * bottom); // bottom ^(3/2)

      //Vector3d f_i_j = r_i_j/ bottom;
      // Resuse r_i_j as f_i_j cause fuck it's verbose otherwise
      r_i_j_x /= bottom;
      r_i_j_y /= bottom;
      r_i_j_z /= bottom;

      // distvector = j pos - i pos
      // particles[i].acceleration accumlator = (m of j/ (dist^2 + e2)) * distvector

      // so f_i_j is the shared part of the calculation between the pair
      // which = distvector/(dist^2 + e2) but I multiply f_i_j by negative 1 to
      // reverse the direction so that it works for particle i too

      // Notice we are just adding to the accelerator
      //particles[i]->setAcceleration(particles[j]->getMass() * f_i_j + particles[i]->getAcceleration());
      //particles[j]->setAcceleration(particles[i]->getMass() * -1 * f_i_j + particles[j]->getAcceleration());
      particles.accelerations.xs[i] += particles.mass[j] * r_i_j_x;
      particles.accelerations.ys[i] += particles.mass[j] * r_i_j_y;
      particles.accelerations.zs[i] += particles.mass[j] * r_i_j_z;

      particles.accelerations.xs[j] -= particles.mass[i] * r_i_j_x;
      particles.accelerations.ys[j] -= particles.mass[i] * r_i_j_y;
      particles.accelerations.zs[j] -= particles.mass[i] * r_i_j_z;
    }
  }
  for (unsigned int i = 0; i < particles.length; ++i) {
    // Integrate with Symplectic euler.
    // Important to update velocity and use updated velocity to update position
    //particles[i]->setVelocity(particles[i]->getVelocity() + h * particles[i]->getAcceleration());
    //particles[i]->setPosition(particles[i]->getPosition() + h * particles[i]->getVelocity());
    particles.velocities.xs[i] += step * particles.accelerations.xs[i];
    particles.velocities.ys[i] += step * particles.accelerations.ys[i];
    particles.velocities.zs[i] += step * particles.accelerations.zs[i];

    // Zero out acceleration now instead because of access patterns? 
    particles.accelerations.xs[i] = 0;
    particles.accelerations.ys[i] = 0;
    particles.accelerations.zs[i] = 0;

    particles.positions.xs[i] += step * particles.velocities.xs[i];
    particles.positions.ys[i] += step * particles.velocities.ys[i];
    particles.positions.zs[i] += step * particles.velocities.zs[i];
  }

}
