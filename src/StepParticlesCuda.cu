#include "hip/hip_runtime.h"
#include <memory>
#include <vector>

#include "Particle.h"

void init(Particles &particles, double step, double softening) {}

void stepParticles(Particles &particles, double step, double softening) {

  //for (unsigned int i = 0; i < particles.length; ++i) {
  //  // Not actually acceleration
  //  // This is the acceleration accumulator
  //  //particles[i]->setAcceleration(Vector3d::Zero());
  //  particles.acceleration[i*3] = 0;
  //  particles.acceleration[i*3 + 1] = 0;
  //  particles.acceleration[i*3 + 2] = 0;
  //}

  for (unsigned int i = 0; i < particles.length; ++i) {
    double xDiff[particles.length];
    double yDiff[particles.length];
    double zDiff[particles.length];
    for (unsigned int j = i + 1; j < particles.length; ++j) {
      xDiff[j] = particles.positions.xs[j] - particles.positions.xs[i];
      yDiff[j] = particles.positions.ys[j] - particles.positions.ys[i];
      zDiff[j] = particles.positions.zs[j] - particles.positions.zs[i];
    }
    // r_i_j is the distance vector
    //Vector3d r_i_j = particles[j]->getPosition() - particles[i]->getPosition();
    //      double r_i_j_x = particles.positions.xs[j] - particles.positions.xs[i];
    //      double r_i_j_y = particles.positions.ys[j] - particles.positions.ys[i];
    //      double r_i_j_z = particles.positions.zs[j] - particles.positions.zs[i];

    for (unsigned int j = i + 1; j < particles.length; ++j) {

      // bottom is scaling factor we divide by, we don't need to separate it out
      //double bottom = r_i_j.squaredNorm() + e2;
      double bottom = xDiff[j] * xDiff[j] + yDiff[j] * yDiff[j] + zDiff[j] * zDiff[j] + softening;
      bottom = sqrt(bottom * bottom * bottom);
      xDiff[j] /= bottom;
      yDiff[j] /= bottom;
      zDiff[j] /= bottom;
    }

    // distvector = j pos - i pos
    // particles[i].acceleration accumlator = (m of j/ (dist^2 + e2)) * distvector

    // so f_i_j is the shared part of the calculation between the pair
    // which = distvector/(dist^2 + e2) but I multiply f_i_j by negative 1 to
    // reverse the direction so that it works for particle i too

    // Notice we are just adding to the accelerator
    //particles[i]->setAcceleration(particles[j]->getMass() * f_i_j + particles[i]->getAcceleration());
    //particles[j]->setAcceleration(particles[i]->getMass() * -1 * f_i_j + particles[j]->getAcceleration());
    for (unsigned int j = i + 1; j < particles.length; ++j) {
      particles.accelerations.xs[i] += particles.mass[j] * xDiff[j];
      particles.accelerations.ys[i] += particles.mass[j] * yDiff[j];
      particles.accelerations.zs[i] += particles.mass[j] * zDiff[j];
    }

    for (unsigned int j = i + 1; j < particles.length; ++j) {
      particles.accelerations.xs[j] -= particles.mass[i] * xDiff[j];
      particles.accelerations.ys[j] -= particles.mass[i] * yDiff[j];
      particles.accelerations.zs[j] -= particles.mass[i] * zDiff[j];
    }
  }
  for (unsigned int i = 0; i < particles.length; ++i) {
    // Integrate with Symplectic euler.
    // Important to update velocity and use updated velocity to update position
    //particles[i]->setVelocity(particles[i]->getVelocity() + h * particles[i]->getAcceleration());
    //particles[i]->setPosition(particles[i]->getPosition() + h * particles[i]->getVelocity());
    particles.velocities.xs[i] += step * particles.accelerations.xs[i];
    particles.velocities.ys[i] += step * particles.accelerations.ys[i];
    particles.velocities.zs[i] += step * particles.accelerations.zs[i];

    // Zero out acceleration now instead because of access patterns? 
    particles.accelerations.xs[i] = 0;
    particles.accelerations.ys[i] = 0;
    particles.accelerations.zs[i] = 0;

    particles.positions.xs[i] += step * particles.velocities.xs[i];
    particles.positions.ys[i] += step * particles.velocities.ys[i];
    particles.positions.zs[i] += step * particles.velocities.zs[i];
  }

}
