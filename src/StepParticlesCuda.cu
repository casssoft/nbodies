#include "hip/hip_runtime.h"
#include <memory>
#include <vector>

#include "Particle.h"


void stepParticles(Particles &particles, double step, double softening) {

  //for (unsigned int i = 0; i < particles.length; ++i) {
  //  // Not actually acceleration
  //  // This is the acceleration accumulator
  //  //particles[i]->setAcceleration(Vector3d::Zero());
  //  particles.acceleration[i*3] = 0;
  //  particles.acceleration[i*3 + 1] = 0;
  //  particles.acceleration[i*3 + 2] = 0;
  //}

  for (unsigned int i = 0; i < particles.length; ++i) {
    for (unsigned int j = i + 1; j < particles.length; ++j) {
      // r_i_j is the distance vector
      //Vector3d r_i_j = particles[j]->getPosition() - particles[i]->getPosition();
      double r_i_j_x = particles.position[j * 3] - particles.position[i * 3];
      double r_i_j_y = particles.position[j * 3 + 1] - particles.position[i * 3 + 1];
      double r_i_j_z = particles.position[j * 3 + 2] - particles.position[i * 3 + 2];

      // bottom is scaling factor we divide by, we don't need to separate it out
      //double bottom = r_i_j.squaredNorm() + e2;
      double bottom = r_i_j_x * r_i_j_x + r_i_j_y * r_i_j_y + r_i_j_z * r_i_j_z + softening;

      bottom = sqrt(bottom * bottom * bottom); // bottom ^(3/2)

      //Vector3d f_i_j = r_i_j/ bottom;
      // Resuse r_i_j as f_i_j cause fuck it's verbose otherwise
      r_i_j_x /= bottom;
      r_i_j_y /= bottom;
      r_i_j_z /= bottom;

      // distvector = j pos - i pos
      // particles[i].acceleration accumlator = (m of j/ (dist^2 + e2)) * distvector

      // so f_i_j is the shared part of the calculation between the pair
      // which = distvector/(dist^2 + e2) but I multiply f_i_j by negative 1 to
      // reverse the direction so that it works for particle i too

      // Notice we are just adding to the accelerator
      //particles[i]->setAcceleration(particles[j]->getMass() * f_i_j + particles[i]->getAcceleration());
      //particles[j]->setAcceleration(particles[i]->getMass() * -1 * f_i_j + particles[j]->getAcceleration());
      particles.acceleration[i * 3] += particles.mass[j] * r_i_j_x;
      particles.acceleration[i * 3 + 1] += particles.mass[j] * r_i_j_y;
      particles.acceleration[i * 3 + 2] += particles.mass[j] * r_i_j_z; 

      particles.acceleration[j * 3] -= particles.mass[i] * r_i_j_x;
      particles.acceleration[j * 3 + 1] -= particles.mass[i] * r_i_j_y; 
      particles.acceleration[j * 3 + 2] -= particles.mass[i] * r_i_j_z; 
    }
  }
  for (unsigned int i = 0; i < particles.length; ++i) {
    // Integrate with Symplectic euler.
    // Important to update velocity and use updated velocity to update position
    //particles[i]->setVelocity(particles[i]->getVelocity() + h * particles[i]->getAcceleration());
    //particles[i]->setPosition(particles[i]->getPosition() + h * particles[i]->getVelocity());
    particles.velocity[i * 3] += step * particles.acceleration[i * 3];
    particles.velocity[i * 3 + 1] += step * particles.acceleration[i * 3 + 1];
    particles.velocity[i * 3 + 2] += step * particles.acceleration[i * 3 + 2];

    // Zero out acceleration now instead because of access patterns? 
    particles.acceleration[i * 3] = 0;
    particles.acceleration[i * 3 + 1] = 0;
    particles.acceleration[i * 3 + 2] = 0;

    particles.position[i * 3] += step * particles.velocity[i * 3];
    particles.position[i * 3 + 1] += step * particles.velocity[i * 3 + 1];
    particles.position[i * 3 + 2] += step * particles.velocity[i * 3 + 2];
  }

}
